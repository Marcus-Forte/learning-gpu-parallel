#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>

__device__ void printDev(float* mem, int length) {
  printf("threadIdx.x: %d\n", threadIdx.x);
  printf("blockDim.x: %d\n", blockDim.x);
  printf("threadIdx.y: %d\n", threadIdx.y);

  int global_index = threadIdx.x + blockDim.x * threadIdx.y;
  
   printf("global index: %d\n", global_index);
  for (int i = 0; i < length; ++i)
  {
    printf("mem %d = %f\n", i, mem[i]);
  }
}

__global__ void printDevMem(float *mem, int length)
{
  printDev(mem, length);
}

int main()
{

  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++)
  {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
  }

  thrust::device_vector<float> D{1, 2, 3};

  float *ptr = thrust::raw_pointer_cast(D.data());

  printDevMem<<<2, 4>>>(ptr, 1);

  wrap_test_print();

  hipDeviceSynchronize();
  return 0;
}